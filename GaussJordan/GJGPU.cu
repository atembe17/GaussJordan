#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include<stdio.h>
#include<math.h>
#include <hip/hip_runtime.h>
#include <>

void ErrorCheck(hipError_t);
//Tile width set according to device 
const int TILE_WIDTH = 32;

//Kernel method to scale diagonal elements to unit value
__global__ void ScaleKernel(float* inpMatrix, unsigned int numberOfRows, unsigned int numberOfColumns, float* outMatrix, int row_consider) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	//Check whether row and col does not exceed the provided values
	if (row < numberOfRows && col < numberOfColumns) {
		//Condition to check if the element is on the diagonal 
		if (row == row_consider && col == row_consider) {
			// If diagonal, divide it by itself
			outMatrix[row_consider * numberOfColumns + row_consider] = inpMatrix[row_consider * numberOfColumns + row_consider]/inpMatrix[row_consider * numberOfColumns + row_consider];
		}
		// Condition to check the row to be considered
		else if (row == row_consider && col != row_consider) {
			//Divide each element of considered row by the diagonal element in that row
			outMatrix[row * numberOfColumns + col] = inpMatrix[row * numberOfColumns + col] / inpMatrix[row_consider*numberOfColumns + row_consider];
		}
	}
}

//Kernel Method to subtract each non diagonal value to make it zero
__global__ void SubtractKernel(float* inpMatrix, unsigned int numberOfRows, unsigned int numberOfColumns, float* outMatrix, int row_consider) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	//Check whether row and col does not exceed the provided values
	if (row < numberOfRows && col < numberOfColumns) {
		if (row!=row_consider) {
			outMatrix[row * numberOfColumns + col] = inpMatrix[row * numberOfColumns + col] - (inpMatrix[row*numberOfColumns + row_consider] * inpMatrix[row_consider*numberOfColumns + col]);
		}
	}
}

//Method to allocate memory and compute the gauss jordan elimination
bool GaussianEliminationGPU(float** matrix, unsigned int numberOfRows, unsigned int numberOfColumns, float** outputMatrix, bool partialPivot) {
	//Variable of type hipError_t to store error status
	hipError_t status;
	//Cuda memory allocation to store the input matrix
	float* Md;
	//Cuda memory allocation for computing matrix on device
	float* Rd;
	//storage in bytes
	int bytes = numberOfRows * numberOfColumns * sizeof(float);
	// Memory allocation
	hipMalloc((void**)&Md, bytes);
	hipMalloc((void**)&Rd, bytes);
	//Check for error
	status = hipGetLastError();
	if (status != hipSuccess)
	{
		ErrorCheck(status);
		hipFree(Md);
		hipFree(Rd);
		return false;
	}
	//Specifying the cuda block size to tile size
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	//Specifying the no of grids
	int grid1 = (int)ceil((float)numberOfColumns / (float)TILE_WIDTH);
	int grid2 = (int)ceil((float)numberOfRows / (float)TILE_WIDTH);
	dim3 dimGrid(grid1,grid2 );
	//Copy contents of each row at a time from host input matrix
	for (int i = 0; i < numberOfRows; i++)
	{
		hipMemcpy(&Md[i*numberOfColumns], matrix[i], numberOfColumns*sizeof(float), hipMemcpyHostToDevice);
	}
	//Memory copy from deviceto device
	hipMemcpy(Rd, Md, bytes, hipMemcpyDeviceToDevice);

	//Consider one row at time for each scaling and Subtraction
	for (int row_consider = 0; row_consider < numberOfRows; row_consider++) {
		//Scale kernel to make diagonal elements 1..
		ScaleKernel << <dimGrid, dimBlock >> > (Md, numberOfRows, numberOfColumns, Rd,row_consider);
		hipDeviceSynchronize();
		//Get error status of last performed function
		status = hipGetLastError();
		if (status != hipSuccess)
		{
			ErrorCheck(status);
			hipFree(Md);
			hipFree(Rd);
			return false;
		}
		//Copy the computed matrix from ScaleKernel method to input matrix
		hipMemcpy(Md, Rd, bytes, hipMemcpyDeviceToDevice);
		//Method to make the elements other than diagonal 0
		SubtractKernel << <dimGrid, dimBlock >> > (Md, numberOfRows, numberOfColumns, Rd, row_consider);
		hipDeviceSynchronize();
		//Get error status of last performed function		
		status = hipGetLastError();
		if (status != hipSuccess)
		{
			ErrorCheck(status);
			hipFree(Md);
			hipFree(Rd);
			return false;
		}
		//Copy the computed matrix from SubtractKernel method to input matrix
		hipMemcpy(Md, Rd, bytes, hipMemcpyDeviceToDevice);
	}

	//Copy contents of each row at a time from device output matrix to host
	for (int i = 0; i < numberOfRows; i++)
	{
		hipMemcpy(outputMatrix[i], &Md[i * numberOfColumns], numberOfColumns * sizeof(float), hipMemcpyDeviceToHost);
	}
	//Free the memory
	hipFree(Md);
	hipFree(Rd);
	return true;
}

//Error detected..Print
void ErrorCheck(hipError_t status) {	
	printf("\nKernel failed!! %s", hipGetErrorString(status));
}